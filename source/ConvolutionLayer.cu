#include "ConvolutionLayer.cuh"

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p,
                hipdnnTensorDescriptor_t input_tensor_desc_p,
                size_t depth_p,
                size_t ker_size,
                size_t stride,
                size_t zp):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        depth(depth_p),
        kernel_size(ker_size),
        filter_stride(stride),
        zero_padding(zp),
        in_C(3),
        out_C(depth_p),
        _randrange(0.01)
{
    checkCudnnErrors( hipdnnCreateFilterDescriptor(&filter_desc) );
    checkCudnnErrors( hipdnnCreateConvolutionDescriptor(&conv_desc) );

    hipdnnDataType_t inp_datatype;
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    const size_t conv_dims = 2;
    const int pad[conv_dims] = {0, 0};
    const int strides[conv_dims] = {filter_stride, filter_stride};
    const int upscale[conv_dims] = {1, 1};

    checkCudnnErrors( hipdnnSetFilter4dDescriptor(filter_desc,
                                                 HIPDNN_DATA_FLOAT,
                                                 HIPDNN_TENSOR_NCHW,
                                                 out_C,
                                                 in_C,
                                                 kernel_size,
                                                 kernel_size) );

    checkCudnnErrors( hipdnnSetConvolutionNdDescriptor(conv_desc,
                                                      conv_dims,
                                                      pad,
                                                      strides,
                                                      upscale,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      HIPDNN_DATA_FLOAT) );

    int tensor_dims = 4;
    int output_tensor_dims[tensor_dims];
    checkCudnnErrors( cudnnGetConvolutionNdForwardOutputDim(conv_desc,
                                                            input_tensor_desc,
                                                            filter_desc,
                                                            tensor_dims,
                                                            output_tensor_dims) );
    out_N = output_tensor_dims[0];
    out_C = output_tensor_dims[1];
    out_H = output_tensor_dims[2];
    out_W = output_tensor_dims[3];

    std::cout << "Conv output forward dims:" << std::endl;
    for (uint i = 0; i < tensor_dims; ++i){
        std::cout << output_tensor_dims[i] << "  ";
    }
    std::cout << std::endl;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&convbias_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(convbias_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, out_C,
                                                 1, 1) );


    checkCudnnErrors( hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                          input_tensor_desc,
                                                          filter_desc,
                                                          conv_desc,
                                                          output_tensor_desc,
                                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                          0,
                                                          &algo) );

    checkCudnnErrors( hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                              input_tensor_desc,
                                                              filter_desc,
                                                              conv_desc,
                                                              output_tensor_desc,
                                                              algo,
                                                              &workspace_size_bytes) );
    //std::cout << "Workspace size: " << workspace_size_bytes << std::endl;

    checkCudaErrors( hipMalloc(&_workspace_forward, workspace_size_bytes) );

    weights_length = in_N * kernel_size * kernel_size * out_C;
    output_length = out_N * out_C * out_H * out_W;

    h_weights = (float*) malloc(sizeof(float) * weights_length);
    h_bias = (float*) malloc(sizeof(float) * out_C);

    checkCudaErrors( hipMalloc(&d_weights, sizeof(float) * weights_length) );
    checkCudaErrors( hipMalloc(&d_bias, sizeof(float) * out_C) );
    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * output_length) );

}



ConvolutionLayer::~ConvolutionLayer() {
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);
    hipdnnDestroyTensorDescriptor(convbias_tensor_desc);
    //TODO: check tensor desc copy

    free(h_weights);
    free(h_bias);

    checkCudaErrors( hipFree(d_weights) );
    checkCudaErrors( hipFree(d_bias) );
    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(_workspace_forward) );
}


void ConvolutionLayer::propagate_forward(float* d_x){
    float alpha = 1.0f;
    float beta = 0.0f;


    checkCudnnErrors( hipdnnConvolutionForward(cudnn_handle,
                                              &alpha,
                                              input_tensor_desc, d_x,
                                              filter_desc, d_weights,
                                              conv_desc, algo,
                                              _workspace_forward, workspace_size_bytes,
                                              &beta,
                                              output_tensor_desc, d_output) );

    checkCudnnErrors( hipdnnAddTensor(cudnn_handle,
                                     &alpha,
                                     convbias_tensor_desc, d_bias,
                                     &alpha,
                                     output_tensor_desc, d_output) );

    /*
    hipdnnTensorDescriptor_t in1;
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&in1) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(in1,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, in_C, in_H, in_W) );


    This may work for classification of 1 example

    hipdnnTensorDescriptor_t out1;
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&out1) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(out1,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, 96, 55, 55) );

    checkCudnnErrors( hipdnnConvolutionForward(cudnn_handle, &alpha, in1,
                                              d_x, filter_desc, d_weights, conv_desc,
                                              algo, _workspace_forward, workspace_size_bytes, &beta,
                                              out1, d_output) );
                                              */
}

void ConvolutionLayer::propagate_backward(float* d_dy, float* d_x) {
    float alpha = 1.0f;
    float beta = 0.0f;


}


void ConvolutionLayer::update_weights(float lr){

}


void ConvolutionLayer::init_weights_random(std::mt19937& gen){
    std::uniform_real_distribution<> get_rand(-_randrange, _randrange);

    for (uint i = 0; i < weights_length; ++i)
        h_weights[i] = static_cast<float>(get_rand(gen));

    for (uint i = 0; i < out_C; ++i)
        h_bias[i] = 1.0f;

    checkCudaErrors( hipMemcpy(d_weights, h_weights,
                                sizeof(float) * weights_length, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_bias, h_bias,
                                sizeof(float) * out_C, hipMemcpyHostToDevice) );
}
