#include "ConvolutionLayer.cuh"

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p,
                hipdnnTensorDescriptor_t input_tensor_desc_p,
                size_t depth_p,
                size_t ker_size,
                size_t stride,
                size_t zp):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        depth(depth_p),
        kernel_size(ker_size),
        filter_stride(stride),
        zero_padding(zp),
        in_C(3),
        out_C(depth_p)
{
    checkCudnnErrors( hipdnnCreateFilterDescriptor(&filter_desc) );
    checkCudnnErrors( hipdnnCreateConvolutionDescriptor(&conv_desc) );

    hipdnnDataType_t inp_datatype;
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    const size_t conv_dims = 2;
    const int pad[conv_dims] = {0, 0};
    const int strides[conv_dims] = {filter_stride, filter_stride};
    const int upscale[conv_dims] = {1, 1};

    checkCudnnErrors( hipdnnSetFilter4dDescriptor(filter_desc,
                                                 HIPDNN_DATA_FLOAT,
                                                 HIPDNN_TENSOR_NCHW,
                                                 out_C,
                                                 in_C,
                                                 kernel_size,
                                                 kernel_size) );

    checkCudnnErrors( hipdnnSetConvolutionNdDescriptor(conv_desc,
                                                      conv_dims,
                                                      pad,
                                                      strides,
                                                      upscale,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      HIPDNN_DATA_FLOAT) );

    int tensor_dims = 4;
    int output_tensor_dims[tensor_dims];
    checkCudnnErrors( cudnnGetConvolutionNdForwardOutputDim(conv_desc,
                                                            input_tensor_desc,
                                                            filter_desc,
                                                            tensor_dims,
                                                            output_tensor_dims) );
    out_N = output_tensor_dims[0];
    out_C = output_tensor_dims[1];
    out_H = output_tensor_dims[2];
    out_W = output_tensor_dims[3];

    std::cout << "Conv output forward dims:" << std::endl;
    for (uint i = 0; i < tensor_dims; ++i){
        std::cout << output_tensor_dims[i] << "  ";
    }
    std::cout << std::endl;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 out_N, out_C,
                                                 out_H, out_W) );


    checkCudnnErrors( hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                          input_tensor_desc,
                                                          filter_desc,
                                                          conv_desc,
                                                          output_tensor_desc,
                                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                          0,
                                                          &algo) );

    checkCudnnErrors( hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                              input_tensor_desc,
                                                              filter_desc,
                                                              conv_desc,
                                                              output_tensor_desc,
                                                              algo,
                                                              &workspace_size_bytes) );
    std::cout << "Workspace size: " << workspace_size_bytes << std::endl;

    checkCudaErrors( hipMalloc(&_workspace, workspace_size_bytes) );
    checkCudaErrors( hipMalloc(&d_weights, sizeof(float) * in_C * kernel_size * kernel_size * out_C) );
    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * in_N * out_C * out_H * out_W) );

}



ConvolutionLayer::~ConvolutionLayer() {
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_weights) );
    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(_workspace) );
}


void ConvolutionLayer::propagate_forward(float* d_x){
    float alpha = 1.0f;
    float beta = 0.0f;

    checkCudnnErrors( hipdnnConvolutionForward(cudnn_handle, &alpha, input_tensor_desc,
                                              d_x, filter_desc, d_weights, conv_desc,
                                              algo, _workspace, workspace_size_bytes, &beta,
                                              output_tensor_desc, d_output) );
}

