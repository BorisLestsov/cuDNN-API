#include "ConvolutionLayer.cuh"

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p,
                hipdnnTensorDescriptor_t data_tensor_desc_p,
                size_t depth_p,
                size_t ker_size,
                size_t stride,
                size_t zp):
        cudnn_handle(cudnn_handle_p),
        depth(depth_p),
        kernel_size(ker_size),
        filter_stride(stride),
        zero_padding(zp),
        in_C(3),
        out_C(3)
{
    checkCudnnErrors( hipdnnCreateFilterDescriptor(&filter_desc) );
    checkCudnnErrors( hipdnnCreateConvolutionDescriptor(&conv_desc) );

    const size_t conv_dims = 2;
    const int pad[conv_dims] = {0, 0};
    const int strides[conv_dims] = {filter_stride, filter_stride};
    const int upscale[conv_dims] = {1, 1};

    checkCudnnErrors( hipdnnSetFilter4dDescriptor(filter_desc,
                                                 HIPDNN_DATA_FLOAT,
                                                 HIPDNN_TENSOR_NCHW,
                                                 out_C,
                                                 in_C,
                                                 kernel_size,
                                                 kernel_size) );

    checkCudnnErrors( hipdnnSetConvolutionNdDescriptor(conv_desc,
                                                      conv_dims,
                                                      pad,
                                                      strides,
                                                      upscale,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      HIPDNN_DATA_FLOAT) );

    int tensor_dims = 4;
    int output_tensor_dims[tensor_dims];
    checkCudnnErrors( cudnnGetConvolutionNdForwardOutputDim(conv_desc,
                                                           data_tensor_desc_p,
                                                           filter_desc,
                                                           tensor_dims,
                                                           output_tensor_dims) );

    for (uint i = 0; i < tensor_dims; ++i){
        std::cout << output_tensor_dims[i] << "    ";
    }
    std::cout << std::endl;
}

/*
ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p):
        cudnn_handle(cudnn_handle_p),

{
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&filter_desc) );
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&conv_desc) );

    checkCudnnErrors(hipdnnSetFilter4dDescriptor(filterDesc,
                                          HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW,
                                          out_C,
                                          in_C,
                                          kernel_size,
                                          kernel_size));

    checkCudnnErrors( hipdnnSetConvolutionNdDescriptor(convDesc,
                                                      convDims,
                                                      pad,
                                                      filterStride,
                                                      upscale,
                                                      CUDNN_CROSS_CORELLATION) );
    checkCudnnErrors(hipdnnGetConvolution2dForwardOutputDim(convDesc,
                                                     srcTensorDesc,
                                                     filterDesc,
                                                     &out_N, &out_C, &out_H, &out_W));
}


ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p,
size_t in_C, size_t out_C, size_t kernel_size,
        size_t in_W, size_t in_H,
        size_t out_W, size_t out_H);
*/


ConvolutionLayer::~ConvolutionLayer() {
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
}
