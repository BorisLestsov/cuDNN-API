#include "hip/hip_runtime.h"
#include "ConvolutionLayer.cuh"

ConvolutionLayer::ConvolutionLayer(hipdnnHandle_t& cudnn_handle_p,
                                   hipblasHandle_t& cublas_handle_p,
                                   hipdnnTensorDescriptor_t input_tensor_desc_p,
                                   size_t depth_p,
                                   size_t ker_size,
                                   size_t stride,
                                   size_t zp):
        cudnn_handle(cudnn_handle_p),
        cublas_handle(cublas_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        depth(depth_p),
        kernel_size(ker_size),
        filter_stride(stride),
        zero_padding(zp),
        in_C(3),
        out_C(depth_p),
        _randrange(0.01)
{
    checkCudnnErrors( hipdnnCreateFilterDescriptor(&filter_desc) );
    checkCudnnErrors( hipdnnCreateConvolutionDescriptor(&conv_desc) );

    hipdnnDataType_t inp_datatype;
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    const size_t conv_dims = 2;
    const int pad[conv_dims] = {0, 0};
    const int strides[conv_dims] = {filter_stride, filter_stride};
    const int upscale[conv_dims] = {1, 1};

    checkCudnnErrors( hipdnnSetFilter4dDescriptor(filter_desc,
                                                 HIPDNN_DATA_FLOAT,
                                                 HIPDNN_TENSOR_NCHW,
                                                 out_C,
                                                 in_C,
                                                 kernel_size,
                                                 kernel_size) );

    checkCudnnErrors( hipdnnSetConvolutionNdDescriptor(conv_desc,
                                                      conv_dims,
                                                      pad,
                                                      strides,
                                                      upscale,
                                                      HIPDNN_CROSS_CORRELATION,
                                                      HIPDNN_DATA_FLOAT) );

    int tensor_dims = 4;
    int output_tensor_dims[tensor_dims];
    checkCudnnErrors( cudnnGetConvolutionNdForwardOutputDim(conv_desc,
                                                            input_tensor_desc,
                                                            filter_desc,
                                                            tensor_dims,
                                                            output_tensor_dims) );
    out_N = output_tensor_dims[0];
    out_C = output_tensor_dims[1];
    out_H = output_tensor_dims[2];
    out_W = output_tensor_dims[3];

    std::cout << "Conv output forward dims:" << std::endl;
    for (uint i = 0; i < tensor_dims; ++i){
        std::cout << output_tensor_dims[i] << "  ";
    }
    std::cout << std::endl;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&bias_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(bias_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, out_C,
                                                 1, 1) );


    checkCudnnErrors( hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                          input_tensor_desc,
                                                          filter_desc,
                                                          conv_desc,
                                                          output_tensor_desc,
                                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                          0,
                                                          &forward_algo) );

    checkCudnnErrors( hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                              input_tensor_desc,
                                                              filter_desc,
                                                              conv_desc,
                                                              output_tensor_desc,
                                                              forward_algo,
                                                              &workspace_size_bytes) );

    size_t tmp_size;
    checkCudnnErrors( hipdnnGetConvolutionBackwardFilterAlgorithm(
            cudnn_handle,
            input_tensor_desc, output_tensor_desc, conv_desc, filter_desc,
            HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &filter_algo) );

    checkCudnnErrors( hipdnnGetConvolutionBackwardFilterWorkspaceSize(
            cudnn_handle,
            input_tensor_desc, output_tensor_desc, conv_desc, filter_desc,
            filter_algo, &tmp_size) );
    if (tmp_size > workspace_size_bytes)
        workspace_size_bytes = tmp_size;

    checkCudnnErrors( hipdnnGetConvolutionBackwardDataAlgorithm(
            cudnn_handle, filter_desc, output_tensor_desc, conv_desc, input_tensor_desc,
            HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &data_algo) );

    checkCudnnErrors( hipdnnGetConvolutionBackwardDataWorkspaceSize(
            cudnn_handle, filter_desc, output_tensor_desc, conv_desc, input_tensor_desc,
            data_algo, &tmp_size) );
    if (tmp_size > workspace_size_bytes)
        workspace_size_bytes = tmp_size;
    // TODO: Use one workspace for all layers

    //std::cout << "Workspace size: " << workspace_size_bytes << std::endl;

    checkCudaErrors( hipMalloc(&d_workspace, workspace_size_bytes) );

    weights_length = in_C * kernel_size * kernel_size * out_C;
    output_length = out_N * out_C * out_H * out_W;
    bias_length = out_C;

    h_weights = (float*) malloc(sizeof(float) * weights_length);
    h_bias = (float*) malloc(sizeof(float) * out_C);

    checkCudaErrors( hipMalloc(&d_weights, sizeof(float) * weights_length) );
    checkCudaErrors( hipMalloc(&d_dweights, sizeof(float) * weights_length) );

    checkCudaErrors( hipMalloc(&d_bias, sizeof(float) * bias_length) );
    checkCudaErrors( hipMalloc(&d_dbias, sizeof(float) * bias_length) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * output_length) );

    checkCudaErrors( hipMalloc(&d_dx, sizeof(float) * in_N * in_C * in_H * in_W) );

}



ConvolutionLayer::~ConvolutionLayer() {
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);
    hipdnnDestroyTensorDescriptor(bias_tensor_desc);
    //TODO: check tensor desc copy

    free(h_weights);
    free(h_bias);

    checkCudaErrors( hipFree(d_workspace) );
    checkCudaErrors( hipFree(d_weights) );
    checkCudaErrors( hipFree(d_dweights) );
    checkCudaErrors( hipFree(d_bias) );
    checkCudaErrors( hipFree(d_dbias) );
    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_dx) );
}


void ConvolutionLayer::propagate_forward(float* d_x){
    float alpha = 1.0f;
    float beta = 0.0f;

    checkCudnnErrors( hipdnnConvolutionForward(cudnn_handle,
                                              &alpha,
                                              input_tensor_desc, d_x,
                                              filter_desc, d_weights,
                                              conv_desc, forward_algo,
                                              d_workspace, workspace_size_bytes,
                                              &beta,
                                              output_tensor_desc, d_output) );

//    float *h_x = (float *) malloc(out_N * out_C * out_H * out_W * sizeof(float));
//    checkCudaErrors(hipMemcpy(h_x, d_output,
//                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));


    checkCudnnErrors( hipdnnAddTensor(cudnn_handle,
                                     &alpha,
                                     bias_tensor_desc, d_bias,
                                     &alpha,
                                     output_tensor_desc, d_output) );

//    checkCudaErrors(hipMemcpy(h_x, d_output,
//                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));

    /*
    hipdnnTensorDescriptor_t in1;
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&in1) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(in1,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, in_C, in_H, in_W) );


    This may work for classification of 1 example

    hipdnnTensorDescriptor_t out1;
    checkCudnnErrors( hipdnnCreateTensorDescriptor(&out1) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(out1,
                                                 HIPDNN_TENSOR_NCHW,
                                                 HIPDNN_DATA_FLOAT,
                                                 1, 96, 55, 55) );

    checkCudnnErrors( hipdnnConvolutionForward(cudnn_handle, &alpha, in1,
                                              d_x, filter_desc, d_weights, conv_desc,
                                              algo, d_workspace, workspace_size_bytes, &beta,
                                              out1, d_output) );
                                              */
}

void ConvolutionLayer::propagate_backward(float* d_dy, float* d_x) {
    float alpha = 1.0f;
    float beta = 0.0f;

    checkCudnnErrors( hipdnnConvolutionBackwardBias(cudnn_handle,
                                                   &alpha,
                                                   output_tensor_desc, d_dy,
                                                   &beta,
                                                   bias_tensor_desc, d_dbias) );


    checkCudnnErrors( hipdnnConvolutionBackwardFilter(cudnn_handle,
                                                     &alpha,
                                                     input_tensor_desc, d_x,
                                                     output_tensor_desc, d_dy,
                                                     conv_desc,
                                                     filter_algo, d_workspace, workspace_size_bytes,
                                                     &beta,
                                                     filter_desc, d_dweights) );

    checkCudnnErrors( hipdnnConvolutionBackwardData(cudnn_handle,
                                                   &alpha,
                                                   filter_desc,
                                                   d_weights, output_tensor_desc, d_dy, conv_desc,
                                                   data_algo, d_workspace, workspace_size_bytes,
                                                   &beta,
                                                   input_tensor_desc, d_dx) );

}


void ConvolutionLayer::update_weights(float lr){
    float alpha = lr;

    checkCublasErrors( hipblasSaxpy(cublas_handle,
                                   weights_length,
                                   &alpha,
                                   d_dweights, 1,
                                   d_weights, 1));

    checkCublasErrors( hipblasSaxpy(cublas_handle,
                                   bias_length,
                                   &alpha,
                                   d_dbias, 1,
                                   d_bias, 1));
}


void ConvolutionLayer::init_weights_random(std::mt19937& gen){
    std::uniform_real_distribution<> get_rand(-_randrange, _randrange);

    for (uint i = 0; i < weights_length; ++i)
        h_weights[i] = static_cast<float>(get_rand(gen));

    for (uint i = 0; i < bias_length; ++i)
        h_bias[i] = 1.0f;

    checkCudaErrors( hipMemcpy(d_weights, h_weights,
                                sizeof(float) * weights_length, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_bias, h_bias,
                                sizeof(float) * bias_length, hipMemcpyHostToDevice) );
}


void ConvolutionLayer::save_kernels(const char* fname){
    std::ofstream f(fname, std::ios::binary | std::ios::out | std::ios::trunc);
    if (!f.good())
        throw std::runtime_error("Could not open file to write kernels");

    f.write((const char*) &in_C, sizeof(int));
    f.write((const char*) &out_C, sizeof(int));
    f.write((const char*) &kernel_size, sizeof(int));
    f.write((const char*) &kernel_size, sizeof(int));

    float *h_x = (float *) malloc(weights_length * sizeof(float));
    checkCudaErrors(hipMemcpy(h_x, d_weights,
                               weights_length * sizeof(float), hipMemcpyDeviceToHost));
    f.write((const char*) h_x, sizeof(float) * weights_length);

    f.close();
}