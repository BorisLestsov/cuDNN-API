#include "hip/hip_runtime.h"
#include "SoftmaxLayer.cuh"

SoftmaxLayer::SoftmaxLayer(hipdnnHandle_t& cudnn_handle_p,
                                 hipdnnTensorDescriptor_t input_tensor_desc_p):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    out_N = in_N;
    out_C = in_C;
    out_H = in_H;
    out_W = in_W;

    n_labels = out_C * out_H * out_W;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );
    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * in_N * in_C * in_H * in_W) );

}

SoftmaxLayer::~SoftmaxLayer() {
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_dx) );
}


void SoftmaxLayer::propagate_forward(float* d_x){
    float alpha = 1.0f, beta = 0.0f;

    float *h_x = (float *) malloc(in_N * in_C * in_H * in_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_x, d_x,
                               in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToHost));




    checkCudnnErrors( hipdnnSoftmaxForward(cudnn_handle,
                                          HIPDNN_SOFTMAX_ACCURATE,
                                          HIPDNN_SOFTMAX_MODE_INSTANCE,
                                          &alpha,
                                          input_tensor_desc, d_x,
                                          &beta,
                                          output_tensor_desc, d_output) );

    float *h_output = (float *) malloc(out_N * out_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_output, d_output,
                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "Softmax:" << std::endl;

    for (uint i = 0; i < out_N; ++i) {
        std::cout << "    EXAMPLE" << std::endl;
        for (uint j = 0; j < out_W; ++j) {
            std::cout << h_output[i*out_W + j] << "    ";
        }
        std::cout << std::endl;
    }
}


void SoftmaxLayer::propagate_backward(float* d_targ, float* d_dx){
    float alpha = 1.0f, beta = 0.0f;

    /*float *h_x = (float *) malloc(in_N * in_C * in_H * in_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_x, d_x,
                               in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToHost));
*/

    // I DONT UNDERSTAND!

    /*int BW = 128;

    checkCudaErrors(hipMemcpyAsync(d_dx,
                                    d_output,
                                    sizeof(float) * out_N * n_labels, hipMemcpyDeviceToDevice));

    compute_softmax_loss<<<_ceil(out_N, BW), BW>>>(d_targ, n_labels, out_N, d_dx);
     */

    /*checkCudnnErrors( hipdnnSoftmaxBackward(cudnn_handle,
                                           HIPDNN_SOFTMAX_ACCURATE,
                                           HIPDNN_SOFTMAX_MODE_INSTANCE,
                                           &alpha,
                                           output_tensor_desc, d_outputs,
                                           output_tensor_desc, d_dy,
                                           input_tensor_desc, d_x,
                                           &beta,
                                           input_tensor_desc, d_dx) );

    float *h_output = (float *) malloc(out_N * out_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_output, d_output,
                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "Softmax:" << std::endl;

    for (uint i = 0; i < out_N; ++i) {
        std::cout << "    EXAMPLE" << std::endl;
        for (uint j = 0; j < out_W; ++j) {
            std::cout << h_output[i*out_W + j] << "    ";
        }
        std::cout << std::endl;
    }
     */
}


__global__ void compute_softmax_loss(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    const int label_value = static_cast<int>(label[idx]);

    // For each item in the batch, decrease the result of the label's value by 1
    diff[idx * num_labels + label_value] -= 1.0f;
}
