#include "SoftmaxLayer.cuh"

SoftmaxLayer::SoftmaxLayer(hipdnnHandle_t& cudnn_handle_p,
                                 hipdnnTensorDescriptor_t input_tensor_desc_p):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    out_N = in_N;
    out_C = in_C;
    out_H = in_H;
    out_W = in_W;

    n_labels = out_C * out_H * out_W;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );
    checkCudaErrors( hipMalloc(&d_dx, sizeof(float) * in_N * in_C * in_H * in_W) );

}

SoftmaxLayer::~SoftmaxLayer() {
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_dx) );
}


void SoftmaxLayer::propagate_forward(float* d_x){
    float alpha = 1.0f, beta = 0.0f;

    checkCudnnErrors( hipdnnSoftmaxForward(cudnn_handle,
                                          HIPDNN_SOFTMAX_ACCURATE,
                                          HIPDNN_SOFTMAX_MODE_INSTANCE,
                                          &alpha,
                                          input_tensor_desc, d_x,
                                          &beta,
                                          output_tensor_desc, d_output) );

//    float *h_output = (float *) malloc(out_N * out_W * sizeof(float));
//    checkCudaErrors(hipMemcpy(h_output, d_output,
//                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));
//    std::cout << "Softmax:" << std::endl;
//
//    for (uint i = 0; i < out_N; ++i) {
//        std::cout << "    EXAMPLE" << std::endl;
//        for (uint j = 0; j < out_W; ++j) {
//            std::cout << h_output[i*out_W + j] << "    ";
//        }
//        std::cout << std::endl;
//    }
}


void SoftmaxLayer::propagate_backward(float* d_dy, float* d_x){
    float alpha = 1.0f, beta = 0.0f;

    /*float *h_x = (float *) malloc(in_N * in_C * in_H * in_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_x, d_x,
                               in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToHost));
*/


    checkCudnnErrors( hipdnnSoftmaxBackward(cudnn_handle,
                                           HIPDNN_SOFTMAX_ACCURATE,
                                           HIPDNN_SOFTMAX_MODE_INSTANCE,
                                           &alpha,
                                           output_tensor_desc, d_output,
                                           output_tensor_desc, d_dy,
                                           &beta,
                                           input_tensor_desc, d_dx) );

}

