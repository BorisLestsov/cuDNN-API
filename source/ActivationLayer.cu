#include "ActivationLayer.cuh"

ActivationLayer::ActivationLayer(hipdnnHandle_t& cudnn_handle_p,
                                 hipdnnTensorDescriptor_t input_tensor_desc_p,
                                 hipdnnActivationMode_t act_f_p):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        act_f(act_f_p)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    out_N = in_N;
    out_C = in_C;
    out_H = in_H;
    out_W = in_W;

    checkCudnnErrors( hipdnnCreateActivationDescriptor(&act_desc) );
    checkCudnnErrors( hipdnnSetActivationDescriptor(act_desc,
                                                   act_f,
                                                   HIPDNN_PROPAGATE_NAN,
                                                   0.0f) );     // TODO: Add clipped relu

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );

}

ActivationLayer::~ActivationLayer() {
    hipdnnDestroyActivationDescriptor(act_desc);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_output) );
}


void ActivationLayer::propagate_forward(float* d_x){
    float alpha = 1.0f, beta = 0.0f;

    checkCudnnErrors( hipdnnActivationForward(cudnn_handle,
                                             act_desc,
                                             &alpha,
                                             input_tensor_desc,
                                             d_x,
                                             &beta,
                                             output_tensor_desc,
                                             d_output) );

}

