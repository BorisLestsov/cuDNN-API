#include "hip/hip_runtime.h"
#include "MSELayer.cuh"

MSELayer::MSELayer(hipdnnHandle_t& cudnn_handle_p,
                   hipdnnTensorDescriptor_t input_tensor_desc_p):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p)

{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    out_N = in_N;
    out_C = 1;
    out_H = 1;
    out_W = 1;

    n_labels = in_C * in_H * in_W;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );
    checkCudaErrors( hipMalloc(&d_dx, sizeof(float) * in_N * in_C * in_H * in_W) );

}

MSELayer::~MSELayer() {
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_dx) );
}


void MSELayer::propagate_forward(float* d_t, float* d_x){

    compute_mse<<<_ceil(in_N, BW), BW>>>(d_t, d_x, n_labels, in_N, d_output);



    float *h_output = (float *) malloc(out_N * out_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_output, d_output,
                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "MSE:" << std::endl;

    float batch_loss = 0.0;
    for (uint i = 0; i < out_N; ++i) {
        batch_loss += h_output[i];
    }

    std::cout << "    Batch loss:" << batch_loss << std::endl;
}


void MSELayer::propagate_backward(float* d_t, float* d_y){

    compute_mse_loss<<<_ceil(out_N, BW), BW>>>(d_t, d_y, n_labels, in_N, d_dx);



//    float *h_output = (float *) malloc(in_N * in_C * in_H * in_W * sizeof(float));
//    checkCudaErrors(hipMemcpy(h_output, d_dx,
//                               in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToHost));
//    std::cout << "MSE:" << std::endl;
//
//    for (uint i = 0; i < in_N; ++i) {
//        std::cout << "    Batch gradient:" << std::endl;
//        for (uint j = 0; j < in_C*in_H*in_W; ++j) {
//            std::cout << h_output[i*n_labels + j] << "    ";
//        }
//        std::cout << std::endl;
//    }
}


__global__ void compute_mse(const float *labels,
                            const float* x,
                            int num_labels,
                            int batch_size,
                            float* losses)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx >= batch_size)
        return;


    int i, stride = batch_idx * num_labels;
    float lbl_diff, loss = 0.0f;
    for (i = 0; i < num_labels; ++i){
        lbl_diff = labels[stride + i] - x[stride + i];
        loss += lbl_diff * lbl_diff;
    }
    losses[batch_idx] = loss;
}


__global__ void compute_mse_loss(const float *labels,
                                 const float* y,
                                 int num_labels,
                                 int batch_size,
                                 float* grad)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx >= batch_size)
        return;


    int i, stride = batch_idx * num_labels;
    for (i = 0; i < num_labels; ++i){
        grad[stride + i] = labels[stride + i] - y[stride + i];
    }
}