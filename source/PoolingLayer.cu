#include "PoolingLayer.cuh"

PoolingLayer::PoolingLayer(hipdnnHandle_t& cudnn_handle_p,
                           hipdnnTensorDescriptor_t input_tensor_desc_p,
                           size_t size_p,
                           size_t stride_p):
        cudnn_handle(cudnn_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        size(size_p),
        stride(stride_p)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    checkCudnnErrors( hipdnnCreatePoolingDescriptor(&pooling_desc) );
    checkCudnnErrors( hipdnnSetPooling2dDescriptor(pooling_desc,
                                                  HIPDNN_POOLING_MAX,
                                                  HIPDNN_PROPAGATE_NAN,
                                                  size, size,
                                                  0, 0,
                                                  stride, stride) );
    checkCudnnErrors( hipdnnGetPooling2dForwardOutputDim(pooling_desc,
                                                        input_tensor_desc,
                                                        &out_N, &out_C, &out_H, &out_W) );

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );

}

PoolingLayer::~PoolingLayer() {
    hipdnnDestroyPoolingDescriptor(pooling_desc);
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    checkCudaErrors( hipFree(d_output) );
}


void PoolingLayer::propagate_forward(float* d_x){
    float alpha = 1.0f, beta = 0.0f;

    float *h_x = (float *) malloc(in_N * in_C * in_H * in_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_x, d_x,
                               in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToHost));

    checkCudnnErrors( hipdnnPoolingForward(cudnn_handle,
                                          pooling_desc,
                                          &alpha,
                                          input_tensor_desc, d_x,
                                          &beta,
                                          output_tensor_desc, d_output) );
}

