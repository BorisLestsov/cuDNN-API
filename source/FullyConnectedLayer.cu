#include "FullyConnectedLayer.cuh"


FullyConnectedLayer::FullyConnectedLayer(hipblasHandle_t& cublas_handle_p, size_t n_inp_p, size_t n_outp_p):
        cublas_handle(cublas_handle_p),
        n_inp(n_inp_p),
        n_outp(n_outp_p)
{
    h_weights = (float*) malloc(n_inp * n_outp * sizeof(float));
    h_bias = (float*) malloc(n_outp * sizeof(float));

    checkCudaErrors( hipMalloc((void**) &d_weights, n_inp * n_outp * sizeof(float)) );
    checkCudaErrors( hipMalloc((void**) &d_bias, n_outp * sizeof(float)) );
}


FullyConnectedLayer::~FullyConnectedLayer() {
    free(h_weights);
    free(h_bias);

    checkCudaErrors( hipFree(d_weights) );
    checkCudaErrors( hipFree(d_bias) );
}


void FullyConnectedLayer::init_weights_random(/* rand function?*/){

    uint i, j;

    for (i = 0; i < n_outp; ++i){
        for (j = 0; j < n_inp; ++j){
            h_weights[i*n_inp + j] = i*n_inp + j;
        }
    }


    for (i = 0; i < n_outp; ++i){
        for (j = 0; j < n_inp; ++j){
            std::cout << h_weights[i*n_inp + j]  << "    ";
        }
        std::cout << std::endl;
    }

    checkCudaErrors( hipMemcpy(d_weights, h_weights,
                                     sizeof(float) * n_inp * n_outp, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_bias, h_bias,
                                     sizeof(float) * n_outp, hipMemcpyHostToDevice) );

}

void FullyConnectedLayer::propagate_forward(float* d_x){
    float alpha = 1.0f;
    float beta = 0.0f;


    /*
     * THis is a working example of Sgevm:

     n_inp = true cols
     n_outp = true rows
    checkCublasErrors( hipblasSgemv(cublas_handle,
                                   HIPBLAS_OP_T,
                                   n_inp, n_outp,
                                   &alpha,
                                   d_weights, n_inp,
                                   d_x, 1,
                                   &beta,
                                   d_x, 1) );
    */

    /*

    Working gemm

    checkCublasErrors( hipblasSgemm(cublas_handle,
                                   HIPBLAS_OP_N, HIPBLAS_OP_N,
                                   colsB, rowsA, colsA
                                   &alpha,
                                   d_B, colsB,
                                   d_A, colsA,
                                   &beta,
                                   d_Store, colsStore) );
    */

}