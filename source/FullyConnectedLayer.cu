#include "FullyConnectedLayer.cuh"


FullyConnectedLayer::FullyConnectedLayer(hipblasHandle_t& cublas_handle_p,
                                         hipdnnTensorDescriptor_t input_tensor_desc_p,
                                         size_t n_outputs_p):
        cublas_handle(cublas_handle_p),
        input_tensor_desc(input_tensor_desc_p),
        n_outp(n_outputs_p),
        _randrange(1.0f)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );

    n_inp = in_C * in_H * in_W;
    out_C = 1;
    out_N = 1;
    out_H = n_outp;
    out_W = in_N;       // in_n == batch_size !

    h_weights = (float*) malloc(n_inp * n_outp * sizeof(float));
    h_bias = (float*) malloc(n_outp * sizeof(float));

    checkCudaErrors( hipMalloc((void**) &d_weights, n_inp * n_outp * sizeof(float)) );
    checkCudaErrors( hipMalloc((void**) &d_bias, n_outp * sizeof(float)) );
    checkCudaErrors( hipMalloc((void**) &d_output, out_H * out_W * sizeof(float)) );

    h_ones = (float*) malloc(out_W * in_N * sizeof(float));
    checkCudaErrors( hipMalloc((void**) &d_ones, out_W * in_N *sizeof(float)) );
    std::fill_n(h_ones, out_W * in_N, 1.0f);
    checkCudaErrors( hipMemcpy(d_ones, h_ones,
                                sizeof(float) * out_W * in_N, hipMemcpyHostToDevice) );
}


FullyConnectedLayer::~FullyConnectedLayer() {
    free(h_weights);
    free(h_bias);
    free(h_ones);

    checkCudaErrors( hipFree(d_weights) );
    checkCudaErrors( hipFree(d_bias) );
    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_ones) );
}


void FullyConnectedLayer::init_weights_random(std::mt19937& gen){

    std::uniform_real_distribution<> get_rand(-_randrange, _randrange);

    size_t weights_length = n_inp * n_outp;
    size_t bias_length = n_outp;

    for (uint i = 0; i < weights_length; ++i)
        h_weights[i] = static_cast<float>(get_rand(gen));
    for (uint i = 0; i < bias_length; ++i)
        h_bias[i] = static_cast<float>(get_rand(gen));

    checkCudaErrors( hipMemcpy(d_weights, h_weights,
                                sizeof(float) * weights_length, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_bias, h_bias,
                                sizeof(float) * bias_length, hipMemcpyHostToDevice) );

}

void FullyConnectedLayer::propagate_forward(float* d_x) {
    float alpha = 1.0f;
    float beta = 0.0f;

    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                n_outp, in_N, n_inp,
                                &alpha,
                                d_weights, n_inp,
                                d_x, n_inp,
                                &beta,
                                d_output, n_outp));

    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                n_outp, in_N, 1,
                                &alpha,
                                d_bias, n_outp,
                                d_ones, 1,
                                &alpha,
                                d_output, n_outp));

}



/*

 Working Test
 void FullyConnectedLayer::propagate_forward(float* d_x) {
    float alpha = 1.0f;
    float beta = 0.0f;

    //
    out_H = 3;
    n_inp = 2;
    n_outp = 3;
    in_N = 4;
    out_W = in_N;


    uint i, j;

    for (i = 0; i < n_outp; ++i) {
        for (j = 0; j < n_inp; ++j) {
            h_weights[i * n_inp + j] = i * n_inp + j;
        }
    }
    for (i = 0; i < n_outp; ++i) {
        h_bias[i] = 1.0f;
    }


    for (i = 0; i < n_outp; ++i) {
        for (j = 0; j < n_inp; ++j) {
            std::cout << h_weights[i * n_inp + j] << "    ";
        }
        std::cout << std::endl;
    }

    float *h_x = (float *) malloc(sizeof(float) * n_inp * in_N);
    for (i = 0; i < in_N; ++i){
        for (j = 0; j < n_inp; ++j){
            h_x[i * n_inp + j] = 2 + i * n_inp + j;
        }
    }
    std::cout << "h_x:" << std::endl;
    for (i = 0; i < in_N; ++i){
        for (j = 0; j < n_inp; ++j){
            std::cout << h_x[i * n_inp + j] << "    ";
        }
        std::cout << std::endl;
    }
    checkCudaErrors(hipMemcpy(d_x, h_x,
                               sizeof(float) * n_inp * in_N, hipMemcpyHostToDevice));


    checkCudaErrors(hipMemcpy(d_weights, h_weights,
                               sizeof(float) * n_inp * n_outp, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_bias, h_bias,
                               sizeof(float) * n_outp, hipMemcpyHostToDevice));

    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                n_outp, in_N, n_inp,
                                &alpha,
                                d_weights, n_inp,
                                d_x, n_inp,
                                &beta,
                                d_output, n_outp));

    checkCublasErrors(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                n_outp, in_N, 1,
                                &alpha,
                                d_bias, n_outp,
                                d_ones, 1,
                                &alpha,
                                d_output, n_outp));

    float *h_output = (float *) malloc(out_H * out_W * sizeof(float));
    checkCudaErrors(hipMemcpy(h_output, d_output,
                               out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));
    std::cout << "res:" << std::endl;
    for (i = 0; i < out_W; ++i){
        for (j = 0; j < out_H; ++j){
            std::cout << h_output[i * out_H + j] << "    ";
        }
        std::cout << std::endl;
    }

}


 */