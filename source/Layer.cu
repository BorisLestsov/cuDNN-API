#include "Layer.cuh"

Layer::Layer(Layer_t type_p, hipdnnTensorDescriptor_t input_tensor_desc,
             hipdnnHandle_t cudnn_handle,
             hipblasHandle_t cublas_handle)
        :
        type(type_p),
        input_tensor_desc(input_tensor_desc),
        cublas_handle(cublas_handle),
        cudnn_handle(cudnn_handle)
{
    int inp_strid;
    checkCudnnErrors( hipdnnGetTensor4dDescriptor(input_tensor_desc,
                                                 &inp_datatype,
                                                 &in_N, &in_C, &in_H, &in_W,
                                                 &inp_strid, &inp_strid, &inp_strid, &inp_strid) );
}


MetricLayer::MetricLayer(Layer_t type_p,
                         hipdnnTensorDescriptor_t input_tensor_desc,
                         hipdnnHandle_t cudnn_handle,
                         hipblasHandle_t cublas_handle):
        Layer(type_p, input_tensor_desc, cudnn_handle, cublas_handle)
{}