#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "helper_functions.h"


int main(){
	hipdnnHandle_t handle;

    checkCUDNN( hipdnnCreate(&handle) );

    checkCUDNN( hipdnnDestroy(handle) );

	return 0;
}