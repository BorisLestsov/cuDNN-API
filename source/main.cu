#include "hip/hip_runtime.h"
#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "helper_functions.h"

__global__ void f() {

}

int main(){
	hipdnnHandle_t handle;

    f<<<1, 1>>>();

    checkCUDNN( hipdnnCreate(&handle) );

    checkCUDNN( hipdnnDestroy(handle) );

	return 0;
}