#include "ConvNet.cuh"


ConvNet::ConvNet(hipdnnHandle_t& cudnn_handle_p,
                 hipblasHandle_t& cublas_handle_p,
                 hipdnnTensorDescriptor_t data_tensor_desc_p,
                 uint seed):

        cudnn_handle(cudnn_handle_p),
        cublas_handle(cublas_handle_p),
        data_tensor_desc(data_tensor_desc_p),


        conv1(cudnn_handle_p, data_tensor_desc_p, 96, 11, 4, 0),
        fc1(cublas_handle_p, conv1.output_tensor_desc, 50),
        act1(cudnn_handle_p, fc1.output_tensor_desc, HIPDNN_ACTIVATION_RELU),
        fc2(cublas_handle_p, act1.output_tensor_desc, 90),
        sm(cudnn_handle_p, fc2.output_tensor_desc),
        nll(cudnn_handle_p, sm.output_tensor_desc),

        gen(seed == 0 ? rd() : seed)
{
    conv1.init_weights_random(gen);
    fc1.init_weights_random(gen);
    fc2.init_weights_random(gen);
}


void ConvNet::fit(TrainData& train, int epoches, float lr){

    for (uint ep = 0; ep < epoches; ++ep) {
        std::cout << "Epoch: " << ep << std::endl;
        while (!train.is_finished()) {
            //std::cout << "Propagating next batch: " << train.get_n_read() << std::endl;

            train.load_next_batch();
            train.copy_batch_to_GPU();


            conv1.propagate_forward(train.d_img_data);
            fc1.propagate_forward(conv1.d_output);
            act1.propagate_forward(fc1.d_output);
            fc2.propagate_forward(act1.d_output);
            sm.propagate_forward(fc2.d_output);
            nll.propagate_forward(train.d_lbl_data, sm.d_output);

            nll.propagate_backward(train.d_lbl_data, sm.d_output);
            sm.propagate_backward(nll.d_dx, fc2.d_output);
            fc2.propagate_backward(sm.d_dx, act1.d_output);
            act1.propagate_backward(fc2.d_dx, fc1.d_output);
            fc1.propagate_backward(act1.d_dx, conv1.d_output);
            conv1.propagate_backward(fc1.d_dx, train.d_img_data);

            conv1.update_weights(lr);
            fc1.update_weights(lr);
            fc2.update_weights(lr);

        }
        train.reset();
    }

}


char* ConvNet::predict(TestData&){
    return nullptr;
}
