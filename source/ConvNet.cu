#include "ConvNet.cuh"


ConvNet::ConvNet(hipdnnHandle_t& cudnn_handle_p,
                 hipblasHandle_t& cublas_handle_p,
                 hipdnnTensorDescriptor_t data_tensor_desc_p,
                 uint seed):

        cudnn_handle(cudnn_handle_p),
        cublas_handle(cublas_handle_p),
        data_tensor_desc(data_tensor_desc_p),


        fc1(cublas_handle_p, data_tensor_desc_p, 90),
        sm(cudnn_handle_p, fc1.output_tensor_desc),

        gen(seed == 0 ? rd() : seed)
{
    fc1.init_weights_random(gen);
}


void ConvNet::fit(TrainData& train){

    /*float* h_dy = (float*) calloc(sm.out_N * sm.out_C * sm.out_H * sm.out_W, sizeof(float));
    float* lbls = (float*) calloc(sm.out_N * sm.out_C * sm.out_H * sm.out_W, sizeof(float));
    lbls[0] = 1.0f;
    lbls[sm.out_C * sm.out_H * sm.out_W] = 1.0f;
*/
    while (!train.is_finished()){
        std::cout << "Propagating next batch: " << train.get_n_read() << std::endl;

        train.load_next_batch();
        train.copy_batch_to_GPU();


        fc1.propagate_forward(train.d_img_data);
        sm.propagate_forward(fc1.d_output);

        sm.propagate_backward(train.d_lbl_data, fc1.d_output);
        fc1.propagate_backward(sm.d_dx, train.d_img_data);
        /*checkCudaErrors(hipMemcpy(d_y, fc1.d_output,
                                   in_N * in_C * in_H * in_W * sizeof(float), hipMemcpyDeviceToDevice));
*/
    }

}


char* ConvNet::predict(TestData&){
    return nullptr;
}
