#include "ConvNet.cuh"


ConvNet::ConvNet(hipdnnHandle_t& cudnn_handle_p,
                 hipblasHandle_t& cublas_handle_p,
                 hipdnnTensorDescriptor_t data_tensor_desc_p,
                 uint seed):

        cudnn_handle(cudnn_handle_p),
        cublas_handle(cublas_handle_p),
        data_tensor_desc(data_tensor_desc_p),

        conv1(cudnn_handle_p, data_tensor_desc_p, 96, 11, 4),
        pool1(cudnn_handle_p, conv1.output_tensor_desc, 2, 2),
        fc1(cublas_handle_p, pool1.output_tensor_desc, 256),
        act1(cudnn_handle_p, fc1.output_tensor_desc, HIPDNN_ACTIVATION_RELU),

        gen(seed == 0 ? rd() : seed)
{
    conv1.init_weights_random(gen);
    fc1.init_weights_random(gen);
}


void ConvNet::fit(TrainData& train){
//    while (!train.is_finished()){
//        std::cout << "Propagating next batch: " << train.get_n_read() << std::endl;
//        train.load_next_batch();
//        conv1.propagate_forward(train.d_img_data);
//        pool1.propagate_forward(conv1.d_output);
//        fc1.propagate_forward(pool1.d_output);
//        /*for (uint i = 0; i < train.loaded; ++i){
//            std::cout << train.ids_data[i] << "   " << train.lbl_data[i] << std::endl;
//        }
//        std::cout << std::endl;*/
//    }
    train.load_next_batch();
    conv1.propagate_forward(train.d_img_data);
    pool1.propagate_forward(conv1.d_output);
    fc1.propagate_forward(pool1.d_output);

    act1.propagate_forward(fc1.d_output);

}


char* ConvNet::predict(TestData&){
    return nullptr;
}
