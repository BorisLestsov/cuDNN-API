#include "hip/hip_runtime.h"
#include "NegLogLikelihoodLayer.cuh"

NegLogLikelihoodLayer::NegLogLikelihoodLayer(hipdnnHandle_t& cudnn_handle_p,
                   hipdnnTensorDescriptor_t input_tensor_desc_p):
        MetricLayer(Layer_t::NLL, input_tensor_desc_p, cudnn_handle_p, nullptr)

{
    out_N = in_N;
    out_C = 1;
    out_H = 1;
    out_W = 1;

    n_labels = in_C * in_H * in_W;

    checkCudnnErrors( hipdnnCreateTensorDescriptor(&output_tensor_desc) );
    checkCudnnErrors( hipdnnSetTensor4dDescriptor(output_tensor_desc,
                                                 HIPDNN_TENSOR_NCHW,
                                                 inp_datatype,
                                                 out_N, out_C,
                                                 out_H, out_W) );

    checkCudaErrors( hipMalloc(&d_output, sizeof(float) * out_N * out_C * out_H * out_W) );
    checkCudaErrors( hipMalloc(&d_dx, sizeof(float) * in_N * in_C * in_H * in_W) );

    h_output = (float*) malloc(sizeof(float) * out_N * out_C * out_H * out_W);
}

NegLogLikelihoodLayer::~NegLogLikelihoodLayer() {
    hipdnnDestroyTensorDescriptor(output_tensor_desc);

    free(h_output);

    checkCudaErrors( hipFree(d_output) );
    checkCudaErrors( hipFree(d_dx) );
}


void NegLogLikelihoodLayer::compute_loss(float *d_t, float *d_x){
#ifdef DEBUG
    std::cout << "nll in: " << cudaCheckNan(d_x, in_N*in_C*in_H*in_W) << std::endl;    
#endif

    compute_nll<<<_ceil(in_N, BW), BW>>>(d_t, d_x, n_labels, in_N, d_output);

    checkCudaErrors(hipMemcpy(h_output, d_output,
                               out_N * out_C * out_H * out_W * sizeof(float), hipMemcpyDeviceToHost));

    batch_loss = 0.0;
    for (uint i = 0; i < out_N; ++i) {
        batch_loss += h_output[i];
    }

#ifdef DEBUG 
    std::cout << "nll out: " << cudaCheckNan(d_output, out_N*out_C*out_H*out_W) << std::endl;
#endif
}


void NegLogLikelihoodLayer::propagate_backward(float* d_t, float* d_y, float momentum){
#ifdef DEBUG
    std::cout << "back nll in: " << cudaCheckNan(d_y, out_N*out_C*out_H*out_W) << std::endl;
    std::cout << "back nll in labels: " << cudaCheckNan(d_t, out_N*out_C*out_H*out_W) << std::endl;
#endif

    compute_nll_grad<<<_ceil(out_N, BW), BW>>>(d_t, d_y, n_labels, in_N, d_dx);


#ifdef DEBUG    
    std::cout << "back nll out: " << cudaCheckNan(d_dx, in_N*in_C*in_H*in_W) << std::endl;
#endif
}





__global__ void compute_nll(const float *labels,
                            const float* x,
                            int num_labels,
                            int batch_size,
                            float* losses)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx >= batch_size)
        return;


    int i, stride = batch_idx * num_labels;
    float loss = 0.0f;
    for (i = 0; i < num_labels; ++i){
        loss += labels[stride + i] * logf(x[stride + i]);
    }
    losses[batch_idx] = -loss;
}


__global__ void compute_nll_grad(const float *labels,
                                 const float* y,
                                 int num_labels,
                                 int batch_size,
                                 float* grad)
{
    int batch_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batch_idx >= batch_size)
        return;


    int i, stride = batch_idx * num_labels;
    for (i = 0; i < num_labels; ++i){
        //printf("%d - %d\n", labels[stride + i], y[stride + i]);
        grad[stride + i] = labels[stride + i] - y[stride + i];
    }
}
